#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <conio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
# include "hip/device_functions.h" 
# include "time.h"
 
#define indis(a,b,c) ( a* b + c) 
#define siraBul(a,b,c,d) ( a + b * c * d)
 
 
 
__global__ void obebi(int buyuk, int kucuk, int *sonuc)
{
	unsigned int i = indis(blockIdx.x,blockDim.x , threadIdx.x); 
    unsigned int j = indis(blockIdx.y,blockDim.y , threadIdx.y);  
	int sira = siraBul(i , j, blockDim.x , gridDim.x);  
	
     __syncthreads();  
	if(sira >kucuk)
		return; 
	// sira > kucuk sayı olunca  dönüyor çünkü  küçük sayı kadar thread var 
 
	if(buyuk % sira == 0 && kucuk % sira == 0) 
	   atomicMax(sonuc,sira);	
	// büyük sayiyla küçük sayinin ortak bölenlerini sırasıyla bulunup yazılıyor
	// bir ortak bölen bulunduğunda  yazılıyor arama devam ettiği sürece tekrar 
	// bir ortak bölen bulunca karşılaştırıp büyük olanı yazdırıyor böylece sonuca ulaşıyoruz
} 
int main()
{   
	int bir, iki;
	printf("1. sayiyi giriniz:" );
	scanf("%d", &bir);
	printf("2. sayiyi giriniz:" );
	scanf("%d", &iki);
 
	int buyuk = 0;
	int kucuk = 0;
 
	if(bir > iki)
	{
		buyuk = bir;
		kucuk = iki;
	}
	else 
	{
		buyuk = iki;
		kucuk = bir;
	}
	 //küçük sayıyı bulmak için kontrol blokları
	int *ay_resim ;	     
 
	int *sonuc = (int*)malloc( sizeof(int));   
 
		int M = 256;
		int N = 256;
 
		if(kucuk <512)
			N = 1 ;
		else 
			N = (int)(kucuk / 512) + 1;  
  
		
		int *ay_sonuc; 
		hipMalloc((void**)&ay_sonuc, sizeof(int));  
		obebi<<<M,N>>>(buyuk, kucuk, ay_sonuc);	 
		hipMemcpy( sonuc, ay_sonuc, sizeof(int), hipMemcpyDeviceToHost);		  
		hipFree(ay_sonuc);
		 //hipMalloc() ile GİB belleği üzerinde yer ayrılmalır.
		//en son olarak ta hipFree() ilebu bellek alanları boşaltılır.
		// <<<m,n>>>ifadesi içerisinde kodun kaç öbek ve kaç iş parçacığı içerisinde icra edileceği görülmektedir. 
		//İş parçacıklarının öbekler içerisinde  bulunduğuna daha önce değinilmişti. 
		//Dolayısıyla toplamda öbek sayısı x iş parçacığı sayısıkadar iş parçacığı üzerinde kod icra edilmiş olmaktadır. 
		//Mesela  <<<10,20>>> ifadesiiçin 10 x 20 = 200 tane iş parçacığı çalışmış olur.
		
		printf("islem sonucu : %d", sonuc[0]); 
		free(sonuc); 
		// sonuc için açılan yer boşaltılıyor
		getch();
 
 
    return 0;
} 
